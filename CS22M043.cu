#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>

using namespace std;

//*******************************************

// Write down the kernels here

__global__ void initializeHP(int* gHp, int T, int H, int* gScore){
    unsigned  int id = blockIdx.x * blockDim.x + threadIdx.x;
    gHp[id] = H;
    gScore[id] = 0;
}

__global__ void intitializeMini(int* gMini, int* gHit, int* gsignal){
    unsigned  int id = blockIdx.x * blockDim.x + threadIdx.x;
    gMini[id] = INT_MAX;
    gHit[id] = 1001;

    if(id == 0) gsignal[0] = 0;
}


__global__ void computeHits(int* gHp, int* gScore, int* gXcoord, int* gYcoord, int round, int T, int* gMini, int* gHit){
    unsigned  int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(gHp[id] <= 0) return;
    int target = (id + round) % T;
    for(int i = 0; i < T; i++){
        if(i == id) continue;
        if(gHp[i] <= 0) continue;

        if(gXcoord[id] == gXcoord[target]){
            if(gXcoord[id] == gXcoord[i]){
                if(gYcoord[id] < gYcoord[target]){
                    if(gYcoord[i] > gYcoord[id] and abs(gYcoord[i] - gYcoord[id]) < gMini[id]){
                        gMini[id] = abs(gYcoord[i] - gYcoord[id]);
                        gHit[id] = i;
                    }
                }
                else{
                    if(gYcoord[i] < gYcoord[id] and abs(gYcoord[i] - gYcoord[id]) < gMini[id]){
                        gMini[id] = abs(gYcoord[i] - gYcoord[id]);
                        gHit[id] = i;
                    }
                }
            }
        }
        else if(gYcoord[id] == gYcoord[target]){
            if(gYcoord[id] == gYcoord[i]){
                if(gXcoord[id] < gXcoord[target]){
                    if(gXcoord[i] > gXcoord[id] and abs(gXcoord[i] - gXcoord[id]) < gMini[id]){
                        gMini[id] = abs(gXcoord[i] - gXcoord[id]);
                        gHit[id] = i;
                    }
                }
                else{
                    if(gXcoord[i] < gXcoord[id] and abs(gXcoord[i] - gXcoord[id]) < gMini[id]){
                        gMini[id] = abs(gXcoord[i] - gXcoord[id]);
                        gHit[id] = i;
                    }
                }
            }
        }
        else{
            if((gYcoord[target] - gYcoord[id]) * (gXcoord[i] - gXcoord[id]) == (gYcoord[i] - gYcoord[id]) * (gXcoord[target] - gXcoord[id])){
                if(gXcoord[id] < gXcoord[target]){
                    if(gXcoord[i] > gXcoord[id] and abs(gXcoord[i] - gXcoord[id]) < gMini[id]){
                        gMini[id] = abs(gXcoord[i] - gXcoord[id]);
                        gHit[id] = i;
                    }
                }
                else{
                    if(gXcoord[i] < gXcoord[id] and abs(gXcoord[i] - gXcoord[id]) < gMini[id]){
                        gMini[id] = abs(gXcoord[i] - gXcoord[id]);
                        gHit[id] = i;
                    }
                }
            }
        }
    }
}


__global__ void updateHP(int* gHp, int* gScore, int* gMini, int* gHit, int* gsignal){
    unsigned  int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(gMini[id] != INT_MAX){
        atomicAdd(&gScore[id], 1);
        atomicAdd(&gHp[gHit[id]], -1);
    }

    __syncthreads();
    if(gHp[id] > 0){
        atomicAdd(&gsignal[0], 1);
    }
}

//***********************************************


int main(int argc,char **argv)
{
    // Variable declarations
    int M,N,T,H,*xcoord,*ycoord,*score;
    

    FILE *inputfilepointer;
    
    //File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer    = fopen( inputfilename , "r");

    if ( inputfilepointer == NULL )  {
        printf( "input.txt file failed to open." );
        return 0; 
    }

    fscanf( inputfilepointer, "%d", &M );
    fscanf( inputfilepointer, "%d", &N );
    fscanf( inputfilepointer, "%d", &T ); // T is number of Tanks
    fscanf( inputfilepointer, "%d", &H ); // H is the starting Health point of each Tank
	
    // Allocate memory on CPU
    xcoord=(int*)malloc(T * sizeof (int));  // X coordinate of each tank
    ycoord=(int*)malloc(T * sizeof (int));  // Y coordinate of each tank
    score=(int*)malloc(T * sizeof (int));  // Score of each tank (ensure that at the end you have copied back the score calculations on the GPU back to this allocation)

    // Get the Input of Tank coordinates
    for(int i=0;i<T;i++)
    {
      fscanf( inputfilepointer, "%d", &xcoord[i] );
      fscanf( inputfilepointer, "%d", &ycoord[i] );
    }
		

    auto start = chrono::high_resolution_clock::now();

    //*********************************
    // Your Code begins here (Do not change anything in main() above this comment)
    //********************************

    int *gHp; //gpu arrays
    int *gScore;
    int *gXcoord;
    int *gYcoord;

    hipMalloc(&gHp, sizeof(int)*(T));
    hipMalloc(&gScore, sizeof(int)*(T));
    hipMalloc(&gXcoord, sizeof(int)*(T));
    hipMalloc(&gYcoord, sizeof(int)*(T));

    int *gMini;
    int *gHit;

    hipMalloc(&gMini, sizeof(int)*(T));
    hipMalloc(&gHit, sizeof(int)*(T));

    hipMemcpy(gScore, score, sizeof(int)*(T), hipMemcpyHostToDevice);

    initializeHP<<<1,T>>>(gHp, T, H, gScore);
    hipDeviceSynchronize();


    
    hipMemcpy(gXcoord, xcoord, sizeof(int)*(T), hipMemcpyHostToDevice);
    hipMemcpy(gYcoord, ycoord, sizeof(int)*(T), hipMemcpyHostToDevice);

    int* gsignal;
    hipMalloc(&gsignal, sizeof(int));

    int round = 1;
    while(1){
        if(round % T == 0){
            round++;
            continue;
        }

        intitializeMini<<<1,T>>>(gMini, gHit, gsignal);
        hipDeviceSynchronize();
        computeHits<<<1, T>>>(gHp, gScore, gXcoord, gYcoord, round, T, gMini, gHit);
        hipDeviceSynchronize();
        
        updateHP<<<1, T>>>(gHp, gScore, gMini, gHit, gsignal);
        hipDeviceSynchronize();

        int* hsignal;
        hsignal = (int*)malloc(sizeof (int));
        hipMemcpy(hsignal, gsignal, sizeof(int), hipMemcpyDeviceToHost);
        
        if(hsignal[0] <= 1)
            break;
        free(hsignal);

        ++round; //last line of the loop
    }

    hipMemcpy(score, gScore, sizeof(int) * (T), hipMemcpyDeviceToHost);

    hipFree(gHp);
    hipFree(gScore);
    hipFree(gXcoord);
    hipFree(gYcoord);
    hipFree(gMini);
    hipFree(gHit);
    hipFree(gsignal);

    //*********************************
    // Your Code ends here (Do not change anything in main() below this comment)
    //********************************

    auto end  = chrono::high_resolution_clock::now();

    chrono::duration<double, std::micro> timeTaken = end-start;

    printf("Execution time : %f\n", timeTaken.count());

    // Output
    char *outputfilename = argv[2];
    char *exectimefilename = argv[3]; 
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename,"w");

    for(int i=0;i<T;i++)
    {
        fprintf( outputfilepointer, "%d\n", score[i]);
    }
    fclose(inputfilepointer);
    fclose(outputfilepointer);

    outputfilepointer = fopen(exectimefilename,"w");
    fprintf(outputfilepointer,"%f", timeTaken.count());
    fclose(outputfilepointer);

    free(xcoord);
    free(ycoord);
    free(score);
    hipDeviceSynchronize();
    return 0;
}