#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>

using namespace std;

//*******************************************

// Write down the kernels here


//***********************************************


int main(int argc,char **argv)
{
    // Variable declarations
    int M,N,T,H,*xcoord,*ycoord,*score;
    

    FILE *inputfilepointer;
    
    //File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer    = fopen( inputfilename , "r");

    if ( inputfilepointer == NULL )  {
        printf( "input.txt file failed to open." );
        return 0; 
    }

    fscanf( inputfilepointer, "%d", &M );
    fscanf( inputfilepointer, "%d", &N );
    fscanf( inputfilepointer, "%d", &T ); // T is number of Tanks
    fscanf( inputfilepointer, "%d", &H ); // H is the starting Health point of each Tank
	
    // Allocate memory on CPU
    xcoord=(int*)malloc(T * sizeof (int));  // X coordinate of each tank
    ycoord=(int*)malloc(T * sizeof (int));  // Y coordinate of each tank
    score=(int*)malloc(T * sizeof (int));  // Score of each tank (ensure that at the end you have copied back the score calculations on the GPU back to this allocation)

    // Get the Input of Tank coordinates
    for(int i=0;i<T;i++)
    {
      fscanf( inputfilepointer, "%d", &xcoord[i] );
      fscanf( inputfilepointer, "%d", &ycoord[i] );
    }
		

    auto start = chrono::high_resolution_clock::now();

    //*********************************
    // Your Code begins here (Do not change anything in main() above this comment)
    //********************************


    //*********************************
    // Your Code ends here (Do not change anything in main() below this comment)
    //********************************

    auto end  = chrono::high_resolution_clock::now();

    chrono::duration<double, std::micro> timeTaken = end-start;

    printf("Execution time : %f\n", timeTaken.count());

    // Output
    char *outputfilename = argv[2];
    char *exectimefilename = argv[3]; 
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename,"w");

    for(int i=0;i<T;i++)
    {
        fprintf( outputfilepointer, "%d\n", score[i]);
    }
    fclose(inputfilepointer);
    fclose(outputfilepointer);

    outputfilepointer = fopen(exectimefilename,"w");
    fprintf(outputfilepointer,"%f", timeTaken.count());
    fclose(outputfilepointer);

    free(xcoord);
    free(ycoord);
    free(score);
    hipDeviceSynchronize();
    return 0;
}